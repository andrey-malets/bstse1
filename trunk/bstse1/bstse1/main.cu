#include "hip/hip_runtime.h"
// Utilities and system includes
#include <shrUtils.h>
#include <iostream>
#include <fstream>
#include <cutil_inline.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <time.h>
using namespace std;



// includes, kernels
#include "kernel.cu"

static char *sSDKsample = "Starting...";

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, float);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

// ��������� ������� ��������� ����� ��� ���������� seed'a warp_standart'a

#define znew   (z=36969*(z&65535)+(z>>16))
#define wnew   (w=18000*(w&65535)+(w>>16))
#define MWC    ((znew<<16)+wnew )
#define SHR3  (jsr^=(jsr<<17), jsr^=(jsr>>13), jsr^=(jsr<<5))
#define CONG  (jcong=69069*jcong+1234567)
#define FIB   ((b=a+b),(a=b-a))
#define KISS  ((MWC^CONG)+SHR3)
#define LFIB4 (c++,t[c]=t[c]+t[UC(c+58)]+t[UC(c+119)]+t[UC(c+178)])
#define SWB   (c++,bro=(x<y),t[c]=(x=t[UC(c+34)])-(y=t[UC(c+19)]+bro))
#define UNI   (KISS*2.328306e-10)
#define VNI   ((long) KISS)*4.656613e-10
#define UC    (unsigned char)  /*a cast operation*/
typedef unsigned long UL;

/*  Global static variables: */
 static UL z=362436069, w=521288629, jsr=123456789, jcong=380116160;
 static UL a=224466889, b=7584631, t[256];
/* Use random seeds to reset z,w,jsr,jcong,a,b, and the table t[256]*/

 static UL x=0,y=0,bro; static unsigned char c=0;

/* Example procedure to set the table, using KISS: */
void settable(UL i1,UL i2,UL i3,UL i4,UL i5, UL i6)
{
	int i; z=i1;w=i2,jsr=i3; jcong=i4; a=i5; b=i6;
	for(i=0;i<256;i=i+1)  t[i]=KISS;
}

void main2(unsigned *res, size_t num)
{
	srand(time(0));
	int a1 = rand();
	int a2 = rand();
	int a3 = rand();
	int a4 = rand();
	int a5 = rand();
	int a6 = rand();

   size_t i;
   settable(a1,a2,a3,a4,a5,a6);
  // settable(1345,6542,3221,123453,651,9118);
	for(i=1; i<num; i++)
	{
		res[i]=KISS;
	} 
}

int main(int argc, char** argv)
{

	printf("[ %s ]\n", sSDKsample);
	char ans;
		do 
		{ 
			runTest(argc, argv);
			printf("Do you want to run again ? Y/N \n");
			cin>>ans;
		}  
		while (ans != 'n');

}


void runTest(int argc, char** argv)
{
	

    // ��������� �������
	size_t
		// ������ ������ � ���������� ������ (� ���������)
		size = 256,
		// ������ ������� � ��������� ������ (� ���������)
		size2 = size * size,
		// ������ ������� � ���������� ������ (� ������)
		bsize = size * sizeof(float),
		// ������ ������� � ��������� ������ (� ������)
		bsize2 = size2 * sizeof(float),
		// ���������� �������� �� �������
		count = 1<<15;
		// ��������� �����
		float dt = 0.005, 
		// ��� �� �������
		m = 0.075,
		// ������������� ����
		d1 = 1;
		// ���������� ����������� ������
		int sn = 1;
		//������ � �������� ����� ������� ��� ���������� � ����������� ���������
		int cif;
		//X ��� Y ������ ���������.
		int XY;
	/*	printf("Enter parametrs of system: Time(dt), Noise(m), Diffusion(d1), Number of saved lines(sn) \n");
		printf("default values 0.005, 0.075, 1, 1 \n");
		scanf("%f%f%f%i", &dt, &m, &d1, &sn);
		*/
		/*cout<<"Enter number of save lines";
		cin>>sn;
		unsigned* h_nsavedata= new unsigned[sn];
		  for(int i=0; i<sn; i++)
			 {
				 cout<<"Enter coordinate for " <<i<<" line ";
				 cin>>cif;
				 h_nsavedata[i]=cif;
			 }*/
		
	  



	    double dt1 = (double) dt;
	    float c1 = (float)0.5*m/pow(dt1,0.5),c2 = (float)0.5*m/pow(dt1,0.5), D = (float) 2*m/pow(dt1, 0.5)*d1;
	
	hipSetDevice(cutGetMaxGflopsDeviceId());
    // ������ ��� ������ ������� ������ ���������
	unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));



	float *h_v = new float[size2], *h_w = new float[size2], *h_stats = new float[count*size*sn], *h_fft = new float[(count/2)*size];

	float *d_v, *d_w, *d_v2, *d_w2, *d_stats;
	unsigned *d_seed, *d_nsavedata;
	hipfftComplex *d_f1;

	cutilSafeCall(hipMalloc((void**) &d_v, bsize2));
	cutilSafeCall(hipMalloc((void**) &d_w, bsize2));
	cutilSafeCall(hipMalloc((void**) &d_v2, bsize2));
	cutilSafeCall(hipMalloc((void**) &d_w2, bsize2));	
	cutilSafeCall(hipMalloc((void**) &d_stats, count * bsize*sn));
//	cutilSafeCall(hipMalloc((void**) &d_nsavedata, sn*sizeof(unsigned)));
	dim3 blockDim(32,32);
	dim3 numBlocks(size/blockDim.x,size/blockDim.y);
	unsigned *h_seed = new unsigned[size2];
	main2(h_seed, size2);

	cutilSafeCall(hipMalloc((void**) &d_seed, size2 * sizeof(unsigned)));
    cutilSafeCall(hipMemcpy(d_seed, h_seed, size2 * sizeof(unsigned), hipMemcpyHostToDevice));
//	cutilSafeCall(hipMemcpy(d_nsavedata, h_nsavedata, sn * sizeof(unsigned), hipMemcpyHostToDevice));
	init <<<numBlocks, blockDim>>>(d_v);
	init <<<numBlocks, blockDim>>>(d_w);

//	RandomGPU<<<numBlocks, threadsPerBlock>>>(2*count, d_v, d_w, d_v2, d_w2, d_stats, size, 1, 1, 0.06, 2, 0.88);
	RandomGPU2<<<numBlocks, blockDim>>>(d_seed, count, d_stats, d_v, d_w, d_v2, d_w2, c1, c2, dt, D, m);
	hipFree(d_seed);
	hipFree(d_v);
    hipFree(d_w);
	hipFree(d_v2);
	hipFree(d_w2);
	
	(cutStopTimer(timer));
    printf("Processing time: %f (ms)\n", cutGetTimerValue( timer));




	cutilCheckError( cutDeleteTimer( timer));
	cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
	
  // cutilSafeCall(hipMemcpy(h_stats, d_stats, count * bsize, hipMemcpyDeviceToHost));
	
	//����� 
	hipfftHandle fftPlan;  
	hipfftComplex *d_fft;
	int numf = size;
	hipfftReal *h = new hipfftReal[(count/2+1)*numf]; 
	float *d_ffta; // ��� ������� ����� ��������������
	hipMalloc((void**)&d_fft,sizeof(hipfftComplex)*(count/2+1)*numf);
	hipMalloc((void**)&d_ffta,sizeof(hipfftReal)*(count/2+1)*numf);
	hipfftPlan1d(&fftPlan, count, HIPFFT_R2C, numf);
	printf("1");
	hipfftExecR2C(fftPlan, d_stats, d_fft);
	printf("2");
	cutilSafeCall( hipDeviceSynchronize() );
	printf("3");
	ComplexAbs <<<count*size/1024,512>>>(d_fft, d_ffta,(count/2+1)*numf);
	cutilSafeCall( hipDeviceSynchronize() );
	printf("4");
	cutilSafeCall(hipMemcpy(h, d_ffta, (count/2+1) * numf*sizeof(float), hipMemcpyDeviceToHost));
	cutilSafeCall( hipDeviceSynchronize() );
    (cutStopTimer(timer));
    printf("Copy device to host: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError(cutDeleteTimer( timer));
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

	{
		char s[1000];
        sprintf(s, "C:\\2D s = %i count = %i dt = %f M = %f d1 = %f.txt", size, count, dt, m, d1);
		std::ofstream output(s);

		//for(int j = 0; j != numf; ++j)
		//{
		//	for(int i = 0; i != count/2+1; ++i)
		//	{
		//		output << h[j*(count/2+1)+i] << "\t";
		//		//output << h[j*(count/2+1)+i]<< "\t";
		//	}
		//	output << std::endl;
		//}
		//
	
		
		hipfftReal *h_fftsum = new hipfftReal[count/2+1];
		for(int i = 0; i != count/2+1; ++i)
			h_fftsum[i]=0;
		

		for(int j = 0; j != numf; ++j)
		{
			for(int i = 0; i != count/2+1; ++i)
				h_fftsum[i] += h[j*(count/2+1)+i];
		}
	/*	output << "size = " << size << "n= " << count << "dt= " << dt << "M= " << m << "d1= " << d1;
		output << std::endl;*/
		for(int i = 0; i != count/2+1; ++i)
		{
		output << h_fftsum[i]/numf << "\t";
			
		}
		output << std::endl;
	}

    printf("Time of extracting data: %f (ms)\n", cutGetTimerValue( timer));
	
    hipDeviceReset();
	
}

